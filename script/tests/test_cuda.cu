
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA kernel to add elements of two arrays
__global__ void add(int *a, int *b, int *c, int N) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N) c[index] = a[index] + b[index];
}

// Function to populate an array with random integers
void random_ints(int* a, int N) {
    for(int i = 0; i < N; i++) {
        a[i] = rand() % 100;
    }
}

int main() {
    int N = 1<<20; // 1 million elements
    int *a, *b, *c; // host copies of a, b, c
    int *d_a, *d_b, *d_c; // device copies of a, b, c
    int size = N * sizeof(int);

    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    // Allocate space for host copies of a, b, c and setup input values
    a = (int *)malloc(size); random_ints(a, N);
    b = (int *)malloc(size); random_ints(b, N);
    c = (int *)malloc(size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    add<<<(N+255)/256, 256>>>(d_a, d_b, d_c, N);

    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Print the first 10 results
    for (int i = 0; i < 10; i++) {
        printf("c[%d] = %d\n", i, c[i]);
    }

    // Cleanup
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    free(a); free(b); free(c);

    return 0;
}
